#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void conv1D(float *N, float *M, float *P, int width, int MASK_WIDTH) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  
    float PValue = 0;
    
    int start_point = tid - (MASK_WIDTH / 2);  

    for (int i = 0; i < MASK_WIDTH; i++) {
        if (start_point + i >= 0 && start_point + i < width) {
            PValue += N[start_point + i] * M[i];  
        }
    }

    if (tid >= 0 && tid < width) {
        P[tid] = PValue;
    }
}

int main() {
    int width, MASK_WIDTH;

    printf("Enter the size of the input array (N): ");
    scanf("%d", &width);
    printf("Enter the size of the convolution kernel (M): ");
    scanf("%d", &MASK_WIDTH);

    int size_N = width * sizeof(float);
    int size_M = MASK_WIDTH * sizeof(float);
    int size_P = width * sizeof(float);
    
    float *h_N = (float*)malloc(size_N);
    float *h_M = (float*)malloc(size_M);
    float *h_P = (float*)malloc(size_P);

    printf("Enter the elements of the input array N (size %d):\n", width);
    for (int i = 0; i < width; i++) {
        printf("N[%d] = ", i);
        scanf("%f", &h_N[i]);
    }

    printf("Enter the elements of the convolution kernel M (size %d):\n", MASK_WIDTH);
    for (int i = 0; i < MASK_WIDTH; i++) {
        printf("M[%d] = ", i);
        scanf("%f", &h_M[i]);
    }

    float *d_N, *d_M, *d_P;
    hipMalloc((void**)&d_N, size_N);
    hipMalloc((void**)&d_M, size_M);
    hipMalloc((void**)&d_P, size_P);

    hipMemcpy(d_N, h_N, size_N, hipMemcpyHostToDevice);
    hipMemcpy(d_M, h_M, size_M, hipMemcpyHostToDevice);

  	dim3 dimGrid(ceil(width/256.0),1,1);
  	dim3 dimBlock(256,1,1);
    conv1D<<<dimGrid, dimBlock>>>(d_N, d_M, d_P, width, MASK_WIDTH);

    hipMemcpy(h_P, d_P, size_P, hipMemcpyDeviceToHost);

    printf("\nConvolution result (P):\n");
    for (int i = 0; i < width; i++) {
        printf("P[%d] = %f\n", i, h_P[i]);
    }

    free(h_N);
    free(h_M);
    free(h_P);
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    return 0;
}
