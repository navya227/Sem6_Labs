#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void countWordOccurrences(char* sentence, char* word, int sentLen, int wordLen, int* count) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx + wordLen > sentLen) return;

    bool match = true;
    for (int i = 0; i < wordLen; i++) {
        if (sentence[idx + i] != word[i]) {
            match = false;
            break;
        }
    }

    if (match) {
        atomicAdd(count, 1);
    }
}

int main() {
    int sentLen, wordLen;

    printf("Enter length of the sentence: ");
    scanf("%d", &sentLen);

    printf("Enter length of the word: ");
    scanf("%d", &wordLen);

    char* h_sentence = (char*)malloc((sentLen) * sizeof(char));
    char* h_word = (char*)malloc((wordLen) * sizeof(char));

    printf("Enter the sentence (no spaces): ");
    scanf("%s", h_sentence);

    printf("Enter the word to count: ");
    scanf("%s", h_word);

    int *h_count = (int*)malloc(sizeof(int));

    char *d_sentence, *d_word;
    int *d_count;

    hipMalloc((void**)&d_sentence, sentLen * sizeof(char));
    hipMalloc((void**)&d_word, wordLen * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(int));

    hipMemcpy(d_sentence, h_sentence, sentLen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, h_word, wordLen * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);

    dim3 blockDim(128);
    dim3 gridDim((sentLen + blockDim.x - 1) / blockDim.x);

    countWordOccurrences<<<gridDim, blockDim>>>(d_sentence, d_word, sentLen, wordLen, d_count);

    hipMemcpy(&h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

    printf("The word \"%s\" appears %d times in the sentence.\n", h_word, h_count);

    free(h_sentence);
    free(h_word);
    hipFree(d_sentence);
    hipFree(d_word);
    hipFree(d_count);

    return 0;
}
