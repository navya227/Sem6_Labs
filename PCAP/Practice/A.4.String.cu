#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void pattern(char* A, int* B, char* res, int* sp, int w){
    int row = threadIdx.x;
    int col = threadIdx.y;
    char letter = A[row * w + col];
    int start = sp[row * w + col];
    int limit = B[row * w + col];

    for(int i = 0; i < limit ; i++){
        res[start + i] = letter;
    }
}

int main(){
    int h, w ;
    printf("Enter h and w : \n");
    scanf("%d %d",&h,&w);
    char *hA = (char*)malloc(h*w*sizeof(char));
    int *hB = (int*)malloc(h*w*sizeof(int));
    int *hsp = (int*)malloc(h*w*sizeof(int));
    
    printf("Enter A : \n");
    for(int i = 0 ; i < h * w ; i++){
        scanf(" %c",&hA[i]);
    }
    
    printf("Enter B : \n");
    for(int i = 0 ; i < h * w ; i++){
        scanf("%d",&hB[i]);
    }
    
    int len = 0;
    for(int i = 0 ; i < h * w ; i++){
        hsp[i] = len;
        len += hB[i];
    }
    
    char *hres = (char*)malloc(len*sizeof(char));
    
    int *dB, *dsp;
    char *dres, *dA;
    hipMalloc((void**)&dA,h*w*sizeof(char));
    hipMalloc((void**)&dB,h*w*sizeof(int));
    hipMalloc((void**)&dsp,h*w*sizeof(int));
    hipMalloc((void**)&dres,len*sizeof(char));
    
    hipMemcpy(dB,hB,h*w*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dsp,hsp,h*w*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dA,hA,h*w*sizeof(char),hipMemcpyHostToDevice);
    
    dim3 dimGrid (1,1,1);
    dim3 dimBlock (h,w,1);
    pattern<<<dimGrid,dimBlock>>>(dA,dB,dres,dsp,w);

    hipMemcpy(hres,dres,len*sizeof(char),hipMemcpyDeviceToHost);
    
    printf("Answer : \n");
    for(int i = 0 ; i<len ; i++){
        printf("%c",hres[i]);
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dres);
    hipFree(dsp);
 
    free(hA);
    free(hB);
    free(hres);
    free(hsp);

}