#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b , int *c, int n){
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    if(threadId < n){
        c[threadId] = a[threadId]+b[threadId];
    }
}

int main(){
    int n = 5;
    int *hA = (int*)malloc(n*sizeof(int));
    int *hB = (int*)malloc(n*sizeof(int));
    int *hC = (int*)malloc(n*sizeof(int));

    for(int i=0;i<n;i++){
        scanf("%d",&hA[i]);
    }
    for(int i=0;i<n;i++){
        scanf("%d",&hB[i]);
    }

    int *dA, *dB, *dC;

    hipMalloc((void**)&dA,n*sizeof(int));
    hipMalloc((void**)&dB,n*sizeof(int));
    hipMalloc((void**)&dC,n*sizeof(int));

    hipMemcpy(dA,hA,n*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, n*sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid (ceil(n/256.0),1,1);
    dim3 dimBlock (256,1,1);

    add<<<dimGrid,dimBlock>>>(dA,dB,dC,n);

    hipMemcpy(hC,dC,n*sizeof(int),hipMemcpyDeviceToHost);

    for(int i=0;i<n;i++){
        printf("%d ",hC[i]);
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    free(hA);
    free(hB);
    free(hC);

    return 0;
}