#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void inclusiveScan(int* input, int* output) {
    __shared__ int temp[10];
    int tid = threadIdx.x;

    temp[tid] = input[tid];
    __syncthreads();

    for (int offset = 1; offset <= tid; offset++) {
        temp[tid] += input[tid - offset];
    }
    __syncthreads();

    output[tid] = temp[tid];
}

int main() {
    int n;
    printf("Enter number of elements: ");
    scanf("%d", &n);

    int* h_input = (int*)malloc(n * sizeof(int));
    int* h_output = (int*)malloc(n * sizeof(int));

    printf("Enter elements: ");
    for (int i = 0; i < n; i++) {
        scanf("%d", &h_input[i]);
    }

    int *d_input, *d_output;
    hipMalloc((void**)&d_input, n * sizeof(int));
    hipMalloc((void**)&d_output, n * sizeof(int));
    hipMemcpy(d_input, h_input, n * sizeof(int), hipMemcpyHostToDevice);

    inclusiveScan<<<1, n>>>(d_input, d_output);

    hipMemcpy(h_output, d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    printf("Inclusive Scan Output: ");
    for (int i = 0; i < n; i++) {
        printf("%d ", h_output[i]);
    }
    printf("\n");

    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);

    return 0;
}
