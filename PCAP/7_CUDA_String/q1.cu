#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024
#define MAX_WORD_LENGTH 100

__global__ void CUDACountWord(char *text, int textLength, char *word, int wordLength, unsigned int *d_count) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only process valid starting positions
    if (i <= textLength - wordLength) {
        bool match = true;
        
        // Compare each character of the word
        for (int j = 0; j < wordLength; j++) {
            if (text[i + j] != word[j]) {
                match = false;
                break;
            }
        }
        
        // If we found a match, increment the counter
        if (match) {
            atomicAdd(d_count, 1);
        }
    }
}

int main() {
    char text[N];
    char word[MAX_WORD_LENGTH];
    char *d_text, *d_word;
    unsigned int count = 0, result;
    unsigned int *d_count;
    
    // Get the input text
    printf("Enter a string: ");
    fgets(text, N, stdin);
    int textLength = strlen(text);
    if (text[textLength - 1] == '\n') text[textLength - 1] = '\0'; // Remove newline
    textLength = strlen(text); // Recalculate length after newline removal
    
    // Get the word to search for
    printf("Enter word to search: ");
    fgets(word, MAX_WORD_LENGTH, stdin);
    int wordLength = strlen(word);
    if (word[wordLength - 1] == '\n') word[wordLength - 1] = '\0'; // Remove newline
    wordLength = strlen(word); // Recalculate length after newline removal
    
    // Timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    // Allocate memory
    hipMalloc((void**)&d_text, textLength * sizeof(char));
    hipMalloc((void**)&d_word, wordLength * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));
    
    // Copy data to device
    hipMemcpy(d_text, text, textLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_word, word, wordLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);
    
    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(error));
    }
    
    // Calculate grid and block dimensions
    // For a simple approach, use 1 thread per potential starting position
    int threadsPerBlock = 256;
    int numBlocks = (textLength + threadsPerBlock - 1) / threadsPerBlock;
    
    // Launch kernel
    CUDACountWord<<<numBlocks, threadsPerBlock>>>(d_text, textLength, d_word, wordLength, d_count);
    
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA ERROR: %s\n", hipGetErrorString(error));
    }
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    // Copy result back to host
    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    printf("Total occurrences of '%s' = %u\n", word, result);
    printf("Time taken: %f ms\n", elapsedTime);
    
    // Free memory
    hipFree(d_text);
    hipFree(d_word);
    hipFree(d_count);
    
    return 0;
}